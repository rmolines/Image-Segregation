#include "hip/hip_runtime.h"
#include <iostream>
#include <queue>
#include <vector>
#include <assert.h>
#include <fstream>
#include "nvgraph.h"
#include <cstdio>
#include <cmath>
#include <algorithm>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "imagem.h"

using namespace std;

typedef std::pair<double, int> custo_caminho;

typedef std::pair<double *, int *> result_sssp;

#define MAX(y,x) (y>x?y:x)    // Calcula valor maximo
#define MIN(y,x) (y<x?y:x)    // Calcula valor minimo

using namespace std;
/* Rotina para somar dois vetores na GPU */ 
__global__ void variance(double *a, double average, double *c, int N) {
    int i=blockIdx.x * blockDim.x + threadIdx.x;
    if (i<N) { 
        c[i] = pow((a[i] - average), 2)/N;
    }
}

__global__ void add_one(int *pixels, int *out, int height, int width) {
    int i=blockIdx.x * blockDim.x + threadIdx.x;
    int j=blockIdx.y * blockDim.y + threadIdx.y;

    double local_avg = 0;
    if (i < height && j < width) {
        for (int k=-1; k<2; k++){   
            for (int l=-1; l<2; l++){
                if (i+k < 16 && i+k > -1 &&
                    j+l < 16 && j+l > -1)
                {    
                    // printf("%d, %d, %d, %d\n", i+k, j+l, (i+k) * width + j+l, (pixels[(i+k) * width + j+l])); 
                    local_avg+=(double)pixels[(i+k) * width + j+l]/9; 
                }
            }
        } 
        // printf("%f\n", local_avg);
        if (pixels[i*width+j] > local_avg){
            out[i*width+j] = 255;
        } else {
            out[i*width+j] = 0;
        }
    }
}

 __global__ void edgeFilter(unsigned char *image_in, unsigned char *image_out, int rowStart, int rowEnd, int colStart, int colEnd)
 {
    int di,dj;
    int i=blockIdx.x * blockDim.x + threadIdx.x;
    int j=blockIdx.y * blockDim.y + threadIdx.y;
    
    for(i = rowStart; i < rowEnd; ++i) 
    {
       for(j = colStart; j < colEnd; ++j) 
       {
          int min = 256;
          int max = 0;
         for(di = MAX(rowStart, i - 1); di <= MIN(i + 1, rowEnd - 1); di++) 
         {
             for(dj = MAX(colStart, j - 1); dj <= MIN(j + 1, colEnd - 1); dj++) 
             {
                if(min>image_in[di*(colEnd-colStart)+dj]) min = image_in[di*(colEnd-colStart)+dj];
                if(max<image_in[di*(colEnd-colStart)+dj]) max = image_in[di*(colEnd-colStart)+dj]; 
             }
         }
         image_out[i*(colEnd-colStart)+j] = max-min;
       }
     }
 }

/* Programa cria dois vetores e soma eles em GPU */
void blur(imagem *img) {
    
    thrust::host_vector<unsigned char> values_cpu(img->total_size);

    for (int i=0; i<values_cpu.size(); i++){
        values_cpu[i] = (int)img->pixels[i];
    }

    thrust::device_vector<unsigned char> values_gpu (values_cpu);

    thrust::device_vector<unsigned char> out_gpu(values_cpu);
  
    dim3 dimGrid(ceil(img->rows/16.0), ceil(img->cols/16.0), 1);
    dim3 dimBlock(16, 16, 1);
    // add_one<<<dimGrid,dimBlock>>>(thrust::raw_pointer_cast(values_gpu.data()), thrust::raw_pointer_cast(out_gpu.data()), img->rows, img->cols);

    edgeFilter<<<dimGrid,dimBlock>>>(thrust::raw_pointer_cast(values_gpu.data()), thrust::raw_pointer_cast(out_gpu.data()), 0, img->rows, 0, img->cols);

    thrust::host_vector<double> new_img (out_gpu);

    for (int i=0; i<new_img.size(); i++){
        img->pixels[i] = new_img[i];
        // cout << new_img[i] << ' ';
    }
   
}

struct compare_custo_caminho {
    bool operator()(custo_caminho &c1, custo_caminho &c2) {
        return c2.first < c1.first;
    }
};

void check(nvgraphStatus_t status) {
    if (status != NVGRAPH_STATUS_SUCCESS) {
        printf("ERROR : %d\n",status);
        exit(0);
    }
}

float *SSSP(imagem *img, vector<int> seeds) {
    std::priority_queue<custo_caminho, std::vector<custo_caminho>, compare_custo_caminho > Q;
    double *custos = new double[img->total_size];
    int *predecessor = new int[img->total_size];
    bool *analisado = new bool[img->total_size];


    // nvgraph setup
    const size_t  n = img->total_size, vertex_numsets = 1, edge_numsets = 1;
    float *sssp_1_h;
    void** vertex_dim;


    // nvgraph variables
    nvgraphStatus_t status; nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSCTopology32I_t CSC_input;
    hipDataType edge_dimT = HIP_R_32F;
    hipDataType* vertex_dimT;
    vector<float> weights_h;
    vector<int> destination_offsets_h;
    vector<int> source_indices_h;
    int offset = 0;

    for (int i = 0; i < img->total_size; i++) {
        predecessor[i] =-1;
        custos[i] = __DBL_MAX__;
        analisado[i] = false;
    }


    for (int i=0; i<img->rows; i++){
        for (int j=0; j<img->cols; j++){
            destination_offsets_h.push_back(offset);

            int vertex = j + i * img->cols;

            
            if (find(begin(seeds), end(seeds), vertex) != end(seeds)) {
                source_indices_h.push_back(img->total_size);
                weights_h.push_back(0.0);
                offset++;
            }

            if (i > 0) {
                int acima = vertex - img->cols;
                double custo_acima = get_edge(img, vertex, acima);
                source_indices_h.push_back(acima);
                weights_h.push_back(custo_acima);
                offset++;
            }

            if (i < img->rows - 1) {
                int abaixo = vertex + img->cols;
                double custo_abaixo = get_edge(img, vertex, abaixo);
                source_indices_h.push_back(abaixo);
                weights_h.push_back(custo_abaixo);
                offset++;
            }


            if (j < img->cols - 1) {
                int direita = vertex + 1;
                double custo_direita = get_edge(img, vertex, direita);
                source_indices_h.push_back(direita);
                weights_h.push_back(custo_direita);
                offset++;
            }

            if (j > 0) {
                int esquerda = vertex - 1;
                double custo_esquerda = get_edge(img, vertex, esquerda);
                source_indices_h.push_back(esquerda);
                weights_h.push_back(custo_esquerda);
                offset++;
            }
        }
    }

    const int nnz = source_indices_h.size();
    destination_offsets_h.push_back(offset);


    // Init host data
    sssp_1_h = (float*)malloc(n*sizeof(float));
    vertex_dim  = (void**)malloc(vertex_numsets*sizeof(void*));
    vertex_dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
    CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
    vertex_dim[0]= (void*)sssp_1_h; vertex_dimT[0] = HIP_R_32F;
    check(nvgraphCreate(&handle));
    check(nvgraphCreateGraphDescr (handle, &graph));
    CSC_input->nvertices = img->total_size; CSC_input->nedges = nnz;
    CSC_input->destination_offsets = &destination_offsets_h[0];
    CSC_input->source_indices = &source_indices_h[0];

    // Set graph connectivity and properties (tranfers)
    check(nvgraphSetGraphStructure(handle, graph, (void*)CSC_input, NVGRAPH_CSC_32));
    check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    check(nvgraphAllocateEdgeData  (handle, graph, edge_numsets, &edge_dimT));
    check(nvgraphSetEdgeData(handle, graph, (void*)&weights_h[0], 0));

    // Solve
    check(nvgraphSssp(handle, graph, 0,  &img->total_size, 0));

    // Get and print result
    check(nvgraphGetVertexData(handle, graph, (void*)sssp_1_h, 0));


    //Clean 
    free(vertex_dim);
    free(vertex_dimT); free(CSC_input);
    check(nvgraphDestroyGraphDescr(handle, graph));
    check(nvgraphDestroy(handle));


    
    
    return sssp_1_h;
}


int main(int argc, char **argv) {
    if (argc < 3) {
        cout << "Uso:  segmentacao_sequencial entrada.pgm saida.pgm\n";
        return -1;
    }

    std::string path(argv[1]);
    std::string path_output(argv[2]);
    imagem *img = read_pgm(path);

    
    int n_fg, n_bg;
    vector<int> seeds_fg, seeds_bg;
    int x, y;
    
    std::cin >> n_fg >> n_bg;

    for (int k = 0; k < n_fg; k++) {
        std::cin >> x >> y;
        int seed_fg = y * img->cols + x;
        seeds_fg.push_back(seed_fg);
    }
     
    for (int k = 0; k < n_bg; k++) {  
        std::cin >> x >> y;
        int seed_bg = y * img->cols + x;
        seeds_bg.push_back(seed_bg);
    }
    
    
    blur(img);

    float *fg = SSSP(img, seeds_fg);
    float *bg = SSSP(img, seeds_bg);
    
    imagem *saida = new_image(img->rows, img->cols);
    
    for (int k = 0; k < saida->total_size; k++) {
        if (fg[k] > bg[k]) {
            saida->pixels[k] = 0;
        } else {
            saida->pixels[k] = 255;
        }
    }
    
    write_pgm(saida, path_output);    
    return 0;
}