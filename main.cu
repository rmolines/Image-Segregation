#include "hip/hip_runtime.h"
#include <iostream>
#include <queue>
#include <vector>
#include <assert.h>
#include <fstream>
#include "nvgraph.h"
#include <cstdio>
#include <cmath>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "imagem.h"

using namespace std;

typedef std::pair<double, int> custo_caminho;

typedef std::pair<double *, int *> result_sssp;

#define MAX(y,x) (y>x?y:x)    // Calcula valor maximo
#define MIN(y,x) (y<x?y:x)    // Calcula valor minimo

using namespace std;


 __global__ void edge_filter(unsigned char *img, unsigned char *out, int rows, int cols)
 {
    int di,dj;
    int i=blockIdx.x * blockDim.x + threadIdx.x;
    int j=blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i< rows && j< cols) {

        int min = 256;
        int max = 0;
        for(di = MAX(0, i - 1); di <= MIN(i + 1, rows - 1); di++) 
        {
            for(dj = MAX(0, j - 1); dj <= MIN(j + 1, cols - 1); dj++) 
            {
            if(min>img[di*cols+dj]) {
                min = img[di*cols+dj];
            }

            if(max<img[di*cols+dj]) { 
                max = img[di*cols+dj]; 
            }
            }
        }
        out[i*cols+j] = max-min;
    }
     
 }

/* Programa cria dois vetores e soma eles em GPU */
void blur(imagem *img) {
    
    thrust::host_vector<unsigned char> values_cpu(img->total_size);

    for (int i=0; i<values_cpu.size(); i++){
        values_cpu[i] = (int)img->pixels[i];
    }

    thrust::device_vector<unsigned char> values_gpu (values_cpu);

    thrust::device_vector<unsigned char> out_gpu(values_cpu);
  
    dim3 dimGrid(ceil(img->rows/16.0), ceil(img->cols/16.0), 1);
    dim3 dimBlock(16, 16, 1);
    // add_one<<<dimGrid,dimBlock>>>(thrust::raw_pointer_cast(values_gpu.data()), thrust::raw_pointer_cast(out_gpu.data()), img->rows, img->cols);

    edge_filter<<<dimGrid,dimBlock>>>(thrust::raw_pointer_cast(values_gpu.data()), thrust::raw_pointer_cast(out_gpu.data()), img->rows, img->cols);

    thrust::host_vector<double> new_img (out_gpu);

    for (int i=0; i<new_img.size(); i++){
        img->pixels[i] = new_img[i];
        // cout << new_img[i] << ' ';
    }
   
}


struct compare_custo_caminho {
    bool operator()(custo_caminho &c1, custo_caminho &c2) {
        return c2.first < c1.first;
    }
};

void check(nvgraphStatus_t status) {
    if (status != NVGRAPH_STATUS_SUCCESS) {
        printf("ERROR : %d\n",status);
        exit(0);
    }
}

float *SSSP(imagem *img, vector<int> seeds) {
    std::priority_queue<custo_caminho, std::vector<custo_caminho>, compare_custo_caminho > Q;
    double *custos = new double[img->total_size];
    int *predecessor = new int[img->total_size];
    bool *analisado = new bool[img->total_size];


    // nvgraph setup
    const size_t  n = img->total_size, vertex_numsets = 1, edge_numsets = 1;
    float *sssp_1_h;
    void** vertex_dim;


    // nvgraph variables
    nvgraphStatus_t status; nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSCTopology32I_t CSC_input;
    hipDataType edge_dimT = HIP_R_32F;
    hipDataType* vertex_dimT;
    vector<float> weights_h;
    vector<int> destination_offsets_h;
    vector<int> source_indices_h;
    int offset = 0;

    for (int i = 0; i < img->total_size; i++) {
        predecessor[i] =-1;
        custos[i] = __DBL_MAX__;
        analisado[i] = false;
    }


    for (int i=0; i<img->rows; i++){
        for (int j=0; j<img->cols; j++){
            destination_offsets_h.push_back(offset);

            int vertex = j + i * img->cols;

            
            if (find(begin(seeds), end(seeds), vertex) != end(seeds)) {
                source_indices_h.push_back(img->total_size);
                weights_h.push_back(0.0);
                offset++;
            }

            if (i > 0) {
                int acima = vertex - img->cols;
                double custo_acima = get_edge(img, vertex, acima);
                source_indices_h.push_back(acima);
                weights_h.push_back(custo_acima);
                offset++;
            }

            if (i < img->rows - 1) {
                int abaixo = vertex + img->cols;
                double custo_abaixo = get_edge(img, vertex, abaixo);
                source_indices_h.push_back(abaixo);
                weights_h.push_back(custo_abaixo);
                offset++;
            }


            if (j < img->cols - 1) {
                int direita = vertex + 1;
                double custo_direita = get_edge(img, vertex, direita);
                source_indices_h.push_back(direita);
                weights_h.push_back(custo_direita);
                offset++;
            }

            if (j > 0) {
                int esquerda = vertex - 1;
                double custo_esquerda = get_edge(img, vertex, esquerda);
                source_indices_h.push_back(esquerda);
                weights_h.push_back(custo_esquerda);
                offset++;
            }
        }
    }

    const int nnz = source_indices_h.size();
    destination_offsets_h.push_back(offset);


    // Init host data
    sssp_1_h = (float*)malloc(n*sizeof(float));
    vertex_dim  = (void**)malloc(vertex_numsets*sizeof(void*));
    vertex_dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
    CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
    vertex_dim[0]= (void*)sssp_1_h; vertex_dimT[0] = HIP_R_32F;
    check(nvgraphCreate(&handle));
    check(nvgraphCreateGraphDescr (handle, &graph));
    CSC_input->nvertices = img->total_size; CSC_input->nedges = nnz;
    CSC_input->destination_offsets = &destination_offsets_h[0];
    CSC_input->source_indices = &source_indices_h[0];

    // Set graph connectivity and properties (tranfers)
    check(nvgraphSetGraphStructure(handle, graph, (void*)CSC_input, NVGRAPH_CSC_32));
    check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    check(nvgraphAllocateEdgeData  (handle, graph, edge_numsets, &edge_dimT));
    check(nvgraphSetEdgeData(handle, graph, (void*)&weights_h[0], 0));

    // Solve
    check(nvgraphSssp(handle, graph, 0,  &img->total_size, 0));

    // Get and print result
    check(nvgraphGetVertexData(handle, graph, (void*)sssp_1_h, 0));


    //Clean 
    free(vertex_dim);
    free(vertex_dimT); free(CSC_input);
    check(nvgraphDestroyGraphDescr(handle, graph));
    check(nvgraphDestroy(handle));


    
    
    return sssp_1_h;
}


int main(int argc, char **argv) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float total_time=0.0, seg_img_time=0.0, sssp_time=0.0, graph_time=0.0;

    if (argc < 3) {
        cout << "Uso:  segmentacao_sequencial entrada.pgm saida.pgm\n";
        return -1;
    }

    std::string path(argv[1]);
    std::string path_output(argv[2]);
    imagem *img = read_pgm(path);

    
    int n_fg, n_bg;
    vector<int> seeds_fg, seeds_bg;
    int x, y;
    
    std::cin >> n_fg >> n_bg;

    for (int k = 0; k < n_fg; k++) {
        std::cin >> x >> y;
        int seed_fg = y * img->cols + x;
        seeds_fg.push_back(seed_fg);
    }
     
    for (int k = 0; k < n_bg; k++) {  
        std::cin >> x >> y;
        int seed_bg = y * img->cols + x;
        seeds_bg.push_back(seed_bg);
    }

    
    hipEventRecord(start);
    cout << "detecting edges..." << endl;
    blur(img);


    cout << "calculating SSSP for fg seeds..." << endl;
    float *fg = SSSP(img, seeds_fg);

    cout << "calculating SSSP for bg seeds..." << endl;
    float *bg = SSSP(img, seeds_bg);


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&graph_time, start, stop); 
    
    
    imagem *saida = new_image(img->rows, img->cols);
    
    cout << "creating new image..." << endl;

    hipEventRecord(start);

    for (int k = 0; k < saida->total_size; k++) {
        if (fg[k] > bg[k]) {
            saida->pixels[k] = 0;
        } else {
            saida->pixels[k] = 255;
        }
    }

 
    write_pgm(saida, path_output);   
       
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&seg_img_time, start, stop); 


    printf("Total time: %fs\n", graph_time/1000+seg_img_time/1000);
    printf("Graph + Solution time: %fs\n", graph_time/1000);
    printf("Image creation time: %fs\n", seg_img_time/1000);


    return 0;
}